// nvcc -arch=sm_90 mma_test.cu -o mma_test
// ./mma_test
// Test passed: all computed values are correct.

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cstdint>

// Macro for CUDA error checking.
#define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(code)
                  << " " << file << " " << line << std::endl;
        if (abort) exit(code);
    }
}

constexpr int M = 16;
constexpr int K = 256;
constexpr int N = 8;

struct FragmentA {
    uint32_t data[M * K / 32];
};

struct FragmentB {
    uint32_t data[N * K / 32];
};

struct FragmentC {
    int data[M * N];
};

__device__ void mma_operator(FragmentC &d, const FragmentA &a, const FragmentB &b, const FragmentC &c) {
    uint32_t const *A = reinterpret_cast<uint32_t const *>(&a);
    uint32_t const *B = reinterpret_cast<uint32_t const *>(&b);
    int const *C = reinterpret_cast<int const *>(&c);
    int *D = reinterpret_cast<int *>(&d);
    asm volatile(
        "mma.sync.aligned.m16n8k256.row.col.s32.b1.b1.s32.and.popc {%0,%1,%2,%3}, "
        "{%4,%5,%6,%7}, "
        "{%8,%9}, {%10,%11,%12,%13};\n"
        : "=r"(D[0]), "=r"(D[1]), "=r"(D[2]), "=r"(D[3])
        : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]),
          "r"(B[0]), "r"(B[1]),
          "r"(C[0]), "r"(C[1]), "r"(C[2]), "r"(C[3])
    );
}

__global__ void mma_kernel(FragmentA *a, FragmentB *b, FragmentC *c, FragmentC *d_out) {
    FragmentC d;
    mma_operator(d, *a, *b, *c);
    *d_out = d;
}

int main() {
    // Allocate and initialize host data.
    FragmentA h_a;
    FragmentB h_b;
    FragmentC h_c = {0}; // bias matrix initialized to zero
    FragmentC h_d_out;

    // Initialize input fragments with all ones.
    for (int i = 0; i < M * K / 32; i++) {
        h_a.data[i] = 0xFFFFFFFF;
    }
    for (int i = 0; i < N * K / 32; i++) {
        h_b.data[i] = 0xFFFFFFFF;
    }
    for (int i = 0; i < M * N; i++) {
        h_c.data[i] = 0;
    }

    // Allocate device memory.
    FragmentA *d_a;
    FragmentB *d_b;
    FragmentC *d_c, *d_out;
    cudaCheckError(hipMalloc(&d_a, sizeof(FragmentA)));
    cudaCheckError(hipMalloc(&d_b, sizeof(FragmentB)));
    cudaCheckError(hipMalloc(&d_c, sizeof(FragmentC)));
    cudaCheckError(hipMalloc(&d_out, sizeof(FragmentC)));

    // Copy data from host to device.
    cudaCheckError(hipMemcpy(d_a, &h_a, sizeof(FragmentA), hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(d_b, &h_b, sizeof(FragmentB), hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(d_c, &h_c, sizeof(FragmentC), hipMemcpyHostToDevice));

    // Launch kernel. (Using 1 block of 32 threads.)
    mma_kernel<<<1, 32>>>(d_a, d_b, d_c, d_out);
    cudaCheckError(hipGetLastError());
    cudaCheckError(hipDeviceSynchronize());

    // Copy the result back to host.
    cudaCheckError(hipMemcpy(&h_d_out, d_out, sizeof(FragmentC), hipMemcpyDeviceToHost));

    // Test correctness:
    // With inputs of all ones and a zero bias, each computed value (from the 256 bit-products)
    // should be the popcount over 256 bits (i.e. 8 * 32 = 256).
    bool test_passed = true;
    const int expected = 256;
    for (int i = 0; i < 4; i++) {
        if (h_d_out.data[i] != expected) {
            std::cerr << "Error: At index " << i << ", expected " 
                      << expected << " but got " << h_d_out.data[i] << std::endl;
            test_passed = false;
        }
    }
    if (test_passed) {
        std::cout << "Test passed: all computed values are correct." << std::endl;
    } else {
        std::cout << "Test failed." << std::endl;
    }

    // Free device memory.
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_out);

    return 0;
}
